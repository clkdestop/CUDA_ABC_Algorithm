#include "hip/hip_runtime.h"
#include "ackley.h"

#if FUNCTION == ACKLEY

__device__ float formulae(float values[])
{	
	return firstCalculation(values) - secondCalculation(values) + 20 + (float) M_E;
}

	__device__ float firstCalculation(float values[])
	{
		float partialResult = 0.0;
		int i;

		for (i=0; i<D; i++)
			partialResult = partialResult + (powf(values[i], 2));
		partialResult = -0.2f * sqrtf(partialResult / D);
		partialResult = -20 * expf(partialResult);
		return partialResult;
	}

	__device__ float secondCalculation(float values[])
	{
		float result = 0.0;
		int i;

		for (i=0; i<D; i++)
			result = result + (cosf(2 * M_PI * values[i]));
		return expf(result / D);
	}

#endif