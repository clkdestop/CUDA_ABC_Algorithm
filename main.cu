#include <time.h>
#include "types.h"
#include "bees.h"
#include "bestBee.h"
#include "bestPosition.h"
#include "printBees.h"
#include "timer.h"
#include "kernel.h"
#include "hip/hip_runtime.h"

int main()
{
	clock_t begin;
	int cycles;
	Bees *dev_bees;
	Bees bees = (Bees) malloc(sizeof (struct bees));
	BestBee bestBee = (BestBee) malloc(sizeof (struct bestBee));

	hipMalloc((void**) &dev_bees, sizeof(Bees));
	
	srand(time(0));
	begin = startTimer();

	setInizializedFalse(bestBee);
	initializeType(bees);
	inizializeBees(bees);

	for (cycles=0; cycles<MAX_CYCLES; cycles++) 
	{
		beesWork(bees);
		//printBees(bees);
		saveBestPosition(bestBee, bees);
		//printBestBee(bestBee);
	}

	printBestBee(bestBee);

	finishTimer(begin);

	hipFree(dev_bees);

	free(bees);
	free(bestBee);
	system("PAUSE");
	return 0;
}