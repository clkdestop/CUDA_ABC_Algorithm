#include "hip/hip_runtime.h"
#include <time.h>
#include "types.h"
#include "bees.h"
#include "bestBee.h"
#include "bestPosition.h"
#include "printBees.h"
#include "timer.h"
#include "kernel.h"
#include "hip/hip_runtime.h"

__global__ void init_curand(hiprandState *state, time_t time) {
	int idx = threadIdx.x;
	hiprand_init(time, idx, 0, &state[idx]);
}

int main()
{
	clock_t begin;
	int cycles;
	Bees dev_bees;
	hiprandState *d_state;
	Bees bees = (Bees) malloc(sizeof (struct bees));
	BestBee bestBee = (BestBee) malloc(sizeof (struct bestBee));

	hipMalloc((void**) &dev_bees, sizeof(Bees));
	hipMalloc(&d_state, SN);

	init_curand<<<1, SN>>>(d_state, time(0));

	srand(time(0));
	begin = startTimer();

	setInizializedFalse(bestBee);
	initializeType(bees);
	hipMemcpy(dev_bees, bees, sizeof(Bees), hipMemcpyHostToDevice);
	inizializeBees(dev_bees);

	for (cycles=0; cycles<MAX_CYCLES; cycles++) 
	{
		beesWork(bees);
		hipMemcpy(bees, dev_bees, sizeof(Bees), hipMemcpyDeviceToHost);
		saveBestPosition(bestBee, bees);
	}

	printBestBee(bestBee);

	finishTimer(begin);

	hipFree(dev_bees);
	hipFree(d_state);

	free(bees);
	free(bestBee);
	system("PAUSE");
	return 0;
}