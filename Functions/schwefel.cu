#include "hip/hip_runtime.h"
#include "schwefel.h"

#if FUNCTION == SCHWEFEL

__device__ float formulae(float values[])
{
	float result = 418.9829f * D;
	return (result) - calculation(values);
}

	__device__ float calculation(float values[])
	{
		float result = 0.0;
		int i;

		for (i=0; i<D; i++)
			result = result + values[i] * sinf(sqrtf(fabs(values[i])));
		return result;
	}

#endif