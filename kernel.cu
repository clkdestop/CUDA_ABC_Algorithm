#include "kernel.h"

void inizializeBees(Bees dev_bees, hiprandState *dev_randState)
{
	cudaEmployedPlacement<<<BLOCK, THREAD_PER_BLOCK>>>(dev_bees, dev_randState);
}



void beesWork(Bees dev_bees, hiprandState *dev_randState)
{
	assignEmployed(dev_bees, dev_randState);
	beesSearch(dev_bees, dev_randState);
}

	void assignEmployed(Bees dev_bees, hiprandState *dev_randState)
	{
		cudaOnlookerPlacement<<<BLOCK, THREAD_PER_BLOCK>>>(dev_bees, dev_randState);
	}

	void beesSearch(Bees dev_bees, hiprandState *dev_randState)
	{
		cudaFoodExploitation<<<BLOCK, THREAD_PER_BLOCK>>>(dev_bees, dev_randState);
	}
