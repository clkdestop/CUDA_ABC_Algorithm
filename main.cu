#include <time.h>
#include "types.h"
#include "bees.h"
#include "bestBee.h"
#include "bestPosition.h"
#include "printBees.h"
#include "timer.h"
#include "kernel.h"
#include "hip/hip_runtime.h"

int main()
{
	clock_t begin;
	int cycles;
	Bees *dev_bees;
	Bees bees = (Bees) malloc(sizeof (struct bees));
	BestBee bestBee = (BestBee) malloc(sizeof (struct bestBee));

	hipMalloc((void**) &dev_bees, sizeof(Bees));
	
	srand(time(0));
	begin = startTimer();

	setInizializedFalse(bestBee);
	initializeType(bees);
	hipMemcpy(dev_bees, bees, sizeof(Bees), hipMemcpyHostToDevice);
	inizializeBees(bees);

	for (cycles=0; cycles<MAX_CYCLES; cycles++) 
	{
		beesWork(bees);
		hipMemcpy(bees, dev_bees, sizeof(Bees), hipMemcpyDeviceToHost);
		saveBestPosition(bestBee, bees);
	}

	printBestBee(bestBee);

	finishTimer(begin);

	hipFree(dev_bees);

	free(bees);
	free(bestBee);
	system("PAUSE");
	return 0;
}