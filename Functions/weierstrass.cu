#include "hip/hip_runtime.h"
#include "weierstrass.h"

#if FUNCTION == WEIERSTRASS

__device__ float formulae(float values[])
{
	double result = D * secondCalculation();
	return firstCalculation(values) - result;
}

	__device__ float firstCalculation(float values[])
	{
		float result = 0.0;
		float num;
		int i;
		for (i=0; i<D; i++)
		{
			num = values[i] + 0.5f;
			result = result + calcolateSummation(num);
		}
		return result;
	}

	__device__ float secondCalculation()
	{
		return calcolateSummation(0.5);
	}

	__device__ float calcolateSummation(float num)
	{
		float result = 0.0;
		float partialResult;
		float a = 0.5;
		float b = 3.0;

		int k;
		int kMax = 20;

		for (k=0; k<kMax; k++) {
			partialResult = 2 * (float) M_PI * powf(b, k) * num;
			result = result + (powf(a, k) * cosf(partialResult));
		}
		return result;
	}

#endif