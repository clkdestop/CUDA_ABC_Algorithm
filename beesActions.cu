#include "hip/hip_runtime.h"
#include "bees.h"
#include "stdio.h"
void initializeType(Bees bees)
{
	int i;
	for (i=0; i<SN; i++)
		chooseBeeType(bees, i);
}

	void chooseBeeType(Bees bees, int i)
	{
		if (i < NUMBER_OF_EMPLOYED)
			setType(bees, i, EMPLOYED);
		else
			setType(bees, i, UNASSIGNED_ONLOOKER);
	}


__global__ void cudaEmployedPlacement(Bees bees, hiprandState *randState)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	employedPlacement(bees, i, randState);
}

	__device__ void employedPlacement(Bees bees, int i, hiprandState *randState)
	{
		float fitness;
	
		if (isEmployed(bees, i)) 
		{
			generateNewPosition(bees, i, randState);
			fitness = evaluateFitness(bees->positions[i]);
			setFitness(bees, i, fitness);
			setTrial(bees, i, 0);
			//setP(bees, i, 0.0);
		}
	}
	
		__device__ BOOL isEmployed(Bees bees, int i)
		{
			return getType(bees, i) == EMPLOYED;
		}

		__device__ void generateNewPosition(Bees bees, int i, hiprandState *randState)
		{
			int y;
			for (y=0; y<D; y++) {
				bees->positions[i][y] = chooseRandomValueBetweenRange(MIN_SEARCH_RANGE, MAX_SEARCH_RANGE, randState);
				//printf("\n%f\n", bees->positions[i][y]);
			}
		}

			__device__ float chooseRandomValueBetweenRange(float lowerBound, float upperBound, hiprandState *randState)
			{
				int tid = threadIdx.x;
				float random = hiprand_uniform(&randState[tid]);;
				float range = upperBound - lowerBound;
				return lowerBound + (random * range);
			}

		__device__ float evaluateFitness(float position[])
		{
			float fitness = formulae(position);
			//if (fitness == 0.0)
				//return 1;
			return fabs(fitness);
		}
		



__global__ void cudaOnlookerPlacement(Bees bees, hiprandState *randState)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	onlookerPlacement(bees, i, randState);
}

	__device__ void onlookerPlacement(Bees bees, int i, hiprandState *randState)
	{
		//generateEmployedP(bees, i);
		if (isUnassignedOnlooker(bees, i))
		{
			chooseOnlookerPosition(bees, i, randState);
			setTrial(bees, i, 0);
			setType(bees, i, ASSIGNED_ONLOOKER);
		}
	}
	/*
		void generateEmployedP(Bees bees, int i)
		{
			float newP;
			if (isEmployed(bees, i)) 
				newP = calcolateP(bees, i);
				setP(bees, i, newP);
		}
	
			float calcolateP(Bees bees, int i)
			{
				float fitnessSummation = 0.0;
				int y;
				for (y=0; y<NUMBER_OF_EMPLOYED; y++)
					fitnessSummation = fitnessSummation + getFitness(bees, i);
				return getFitness(bees, i) / fitnessSummation;
			}
	*/
		__device__ BOOL isUnassignedOnlooker(Bees bees, int i)
		{
			return getType(bees, i) == UNASSIGNED_ONLOOKER;
		}

		__device__ void chooseOnlookerPosition(Bees bees, int i, hiprandState *randState)
		{
			int selectedEmployed = tournamentEmployedSelection(bees, randState);
			moveOnlookerInPosition(bees, i, selectedEmployed);
		}
	/*
			int rouletteWheelEmployedSelection(Bees bees)
			{
				float totalFitness = getFitness(bees, 0);
				int y;
				for (y=1; y<NUMBER_OF_EMPLOYED; y++)
				{
					totalFitness = totalFitness + getFitness(bees, y);
					if (chooseRandomValueBetweenRange(0.0f, 1.0f) < getFitness(bees, y) / totalFitness)
						return y;
				}
				return 0;
			}
	*/
			__device__ int tournamentEmployedSelection(Bees bees, hiprandState *randState)
			{
				int tournamentIndex[calcolateTournamentSize()];
				int i;
			
				for (i=0; i<calcolateTournamentSize(); i++)
					tournamentIndex[i] = chooseRandomValueBetweenRange(0, NUMBER_OF_EMPLOYED, randState);
					//tournamentIndex[i] = (rand() % (NUMBER_OF_EMPLOYED - 0)) + 0;

				return winnerTournament(bees, tournamentIndex);
			}

				__device__ int winnerTournament(Bees bees, int tournamentIndex[])
				{
					int i;
					int winnerBee = tournamentIndex[0];
					double winnerFitness = getFitness(bees, tournamentIndex[0]);

					for (i=1; i<calcolateTournamentSize(); i++)
					{
						if (getFitness(bees, tournamentIndex[i]) < winnerFitness)
						{
							winnerFitness = getFitness(bees, tournamentIndex[i]);
							winnerBee = tournamentIndex[i];
						}
					}
					return winnerBee;
				}

			__device__ void moveOnlookerInPosition(Bees bees, int i, int selectedEmployed)
			{
				setPosition(bees, i, getPosition(bees, selectedEmployed));
				setFitness(bees, i, getFitness(bees, selectedEmployed));
			}
	


__global__ void cudaFoodExploitation(Bees bees, hiprandState *randState)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	foodExploitation(bees, i, randState);
}

	__device__ void foodExploitation(Bees bees, int i, hiprandState *randState)
	{
		if (hasExceededTheLimit(bees, i))
			resetBee(bees, i, randState);
		else
			tryToFindBetterPosition(bees, i, randState);
	}

		__device__ BOOL hasExceededTheLimit(Bees bees, int i)
		{
			return getTrial(bees, i) > LIMIT;
		}


		__device__ void resetBee(Bees bees, int i, hiprandState *randState)
		{
			if (isEmployed(bees, i))
				employedPlacement(bees, i, randState);
			else
				setType(bees, i, UNASSIGNED_ONLOOKER);
		}

		__device__ void tryToFindBetterPosition(Bees bees, int i, hiprandState *randState)
		{
			float perturbedPosition[D];
			generatePerturbedPosition(bees, i, perturbedPosition, randState);
			chooseBestPosition(bees, i, perturbedPosition);
		}

		__device__ void generatePerturbedPosition(Bees bees, int i, float perturbedPosition[], hiprandState *randState)
		{
			int y, k;
			for (y=0; y<D; y++)
			{
				k = chooseIndex(i, randState);
				perturbedPosition[y] = bees->positions[i][y] + chooseRandomValueBetweenRange(-1.0, 1.0, randState) * (bees->positions[i][y] - bees->positions[k][y]);
				controlifExceedSearchField(perturbedPosition, y);
			}
		}

			__device__ int chooseIndex(int i, hiprandState *randState)
			{
				int index;
				do
					//index = (rand() % (SN - 0)) + 0;
					index = chooseRandomValueBetweenRange(0, SN, randState);
				while (index == i);

				return index;
			}

			__device__ void controlifExceedSearchField(float newPosition[], int y)
			{
				if (newPosition[y] > MAX_SEARCH_RANGE)
					newPosition[y] = MAX_SEARCH_RANGE;
				else if (newPosition[y] < MIN_SEARCH_RANGE)
					newPosition[y] = MIN_SEARCH_RANGE;
			}

		__device__ void chooseBestPosition(Bees bees, int i, float perturbedPosition[])
		{
			float perturbedFitness = evaluateFitness(perturbedPosition);

			if (isPerturbedFitnessBetter(bees, i, perturbedFitness))
				replacePosition(bees, i, perturbedPosition, perturbedFitness);	
			else
				increaseTrial(bees, i);
		}

			__device__ BOOL isPerturbedFitnessBetter(Bees bees, int i, float perturbedFitness)
			{
				return getFitness(bees, i) > perturbedFitness;
			}

			__device__ void replacePosition(Bees bees, int i, float perturbedPosition[], float perturbedFitness)
			{
				setPosition(bees, i, perturbedPosition);
				setFitness(bees, i, perturbedFitness);
				setTrial(bees, i, 0);
			}

			__device__ void increaseTrial(Bees bees, int i)
			{
				int trial = getTrial(bees, i);
				setTrial(bees, i, trial+1);
			}
