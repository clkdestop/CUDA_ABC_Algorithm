#include "hip/hip_runtime.h"
#include "schwefel.h"

#if FUNCTION == SCHWEFEL

float formulae(float values[])
{
	float result = 418.9829f * D;
	return (result) - calculation(values);
}

	float calculation(float values[])
	{
		float result = 0.0;
		int i;

		for (i=0; i<D; i++)
			result = result + values[i] * sinf(sqrtf(fabs(values[i])));
		return result;
	}

#endif