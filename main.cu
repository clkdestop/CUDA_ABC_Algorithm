#include "hip/hip_runtime.h"
#include <time.h>
#include "types.h"
#include "bees.h"
#include "bestBee.h"
#include "bestPosition.h"
#include "printBees.h"
#include "timer.h"
#include "kernel.h"
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"

__global__ void init_curand(hiprandState *randState, time_t time) {
	int tid = threadIdx.x;
	hiprand_init(time, tid, 0, &randState[tid]);
}

int main()
{
	clock_t begin;
	int cycles;
	Bees dev_bees;
	hiprandState *dev_randState;
	Bees bees = (Bees) malloc(sizeof (struct bees));
	BestBee bestBee = (BestBee) malloc(sizeof (struct bestBee));

	hipMalloc((void**) &dev_bees, sizeof(Bees));
	hipMalloc(&dev_randState, SN);

	init_curand<<<1, SN>>>(dev_randState, time(0));

	srand(time(0));
	begin = startTimer();

	setInizializedFalse(bestBee);
	initializeType(bees);
	hipMemcpy(dev_bees, bees, sizeof(Bees), hipMemcpyHostToDevice);
	inizializeBees(dev_bees, dev_randState);

	for (cycles=0; cycles<MAX_CYCLES; cycles++) 
	{
		beesWork(bees);
		hipMemcpy(bees, dev_bees, sizeof(Bees), hipMemcpyDeviceToHost);
		saveBestPosition(bestBee, bees);
	}

	printBestBee(bestBee);

	finishTimer(begin);

	hipFree(dev_bees);
	hipFree(dev_randState);

	free(bees);
	free(bestBee);
	system("PAUSE");
	return 0;
}