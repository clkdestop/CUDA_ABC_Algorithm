#include "kernel.h"

void inizializeBees(Bees dev_bees, hiprandState *dev_randState)
{
	/*int i;
	for (i=0; i<NUMBER_OF_EMPLOYED; i++)
		employedPlacement(bees, i);*/
	cudaEmployedPlacement<<<1, SN>>>(dev_bees, dev_randState);
}


/*
void beesWork(Bees bees)
{
	assignEmployed(bees);
	beesSearch(bees);
}

	void assignEmployed(Bees bees)
	{
		int i;
		for (i=NUMBER_OF_ONLOOKER; i<SN; i++)
			onlookerPlacement(bees, i);
	}

	void beesSearch(Bees bees)
	{
		int i;
		for (i=0; i<SN; i++)
			foodExploitation(bees, i);
	}
*/