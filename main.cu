#include "hip/hip_runtime.h"
#include <time.h>
#include "types.h"
#include "bees.h"
#include "bestBee.h"
#include "bestPosition.h"
#include "printBees.h"
#include "timer.h"
#include "kernel.h"
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"

__global__ void init_curand(hiprandState *randState, time_t time) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(time, tid, 0, &randState[tid]);
}

int main()
{
	clock_t begin;
	int cycles;
	//float elapsedTime;
	Bees dev_bees;
	hiprandState *dev_randState;
	//hipEvent_t start, stop;

	Bees bees = (Bees) malloc(sizeof (struct bees));
	BestBee bestBee = (BestBee) malloc(sizeof (struct bestBee));

	hipMalloc((void**) &dev_bees, sizeof(struct bees));
	hipMalloc(&dev_randState, SN);

	init_curand<<<BLOCK, THREAD_PER_BLOCK>>>(dev_randState, time(0));

	srand((unsigned int) time(0));
	begin = startTimer();
	/*hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0 );*/

	setInizializedFalse(bestBee);
	initializeType(bees);
	hipMemcpy(dev_bees, bees, sizeof(struct bees), hipMemcpyHostToDevice);
	inizializeBees(dev_bees, dev_randState);

	//printBees(bees);

	for (cycles=0; cycles<MAX_CYCLES; cycles++) 
	{
		beesWork(dev_bees, dev_randState);
		//hipMemcpy(bees, dev_bees, sizeof(struct bees), hipMemcpyDeviceToHost);
		//saveBestPosition(bestBee, bees);
		//printBees(bees);
	}
	
	//printBestBee(bestBee);

	finishTimer(begin);
	/*hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	
	hipEventElapsedTime( &elapsedTime, start, stop );
	printf( "Time to generate: %3.1f s\n", elapsedTime  / CLOCKS_PER_SEC);
	hipEventDestroy( start );
	hipEventDestroy( stop );*/

	hipFree(dev_bees);
	hipFree(dev_randState);

	free(bees);
	free(bestBee);
	system("PAUSE");
	return 0;
}