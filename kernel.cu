#include "kernel.h"

void inizializeBees(Bees dev_bees, hiprandState *dev_randState)
{
	/*int i;
	for (i=0; i<NUMBER_OF_EMPLOYED; i++)
		employedPlacement(bees, i);*/
	cudaEmployedPlacement<<<1, SN>>>(dev_bees, dev_randState);
}



void beesWork(Bees dev_bees, hiprandState *dev_randState)
{
	assignEmployed(dev_bees, dev_randState);
	beesSearch(dev_bees, dev_randState);
}

	void assignEmployed(Bees dev_bees, hiprandState *dev_randState)
	{
		/*int i;
		for (i=NUMBER_OF_ONLOOKER; i<SN; i++)
			onlookerPlacement(bees, i);*/
		cudaOnlookerPlacement<<<1, SN>>>(dev_bees, dev_randState);
	}

	void beesSearch(Bees dev_bees, hiprandState *dev_randState)
	{
		/*int i;
		for (i=0; i<SN; i++)
			foodExploitation(bees, i);*/
		cudaFoodExploitation<<<1, SN>>>(dev_bees, dev_randState);
	}
