#include "hip/hip_runtime.h"
#include "nonContinuousRastrigin.h"

#if FUNCTION == NON_CONTINUOUS_RASTRIGIN

__device__ float formulae(float values[])
{
	float result = 0.0;
	float partialResult, value;
	int i;

	for (i=0; i<D; i++)
	{
		value = calculateValue(values[i]);
		partialResult = 2 * (float) M_PI * value;
		partialResult = -10 * cosf(partialResult);
		partialResult = partialResult + 10;
		result = result + powf(value, 2) + partialResult;
	}
	return result;
}

	__device__ float calculateValue(float value)
	{
		if (fabs(value) >= 0.5)
			return (2 * (float) round2(value)) / 2;
		return value;
	}

		__device__ int round2(float number)
		{
			return (number >= 0) ? (int)(number + 0.5) : (int)(number - 0.5);
		}

#endif